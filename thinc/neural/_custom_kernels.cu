#include "hip/hip_runtime.h"
extern "C" __global__
void sum_pool(float* output,
    const float* X, const int* lengths, int B, int T, int O)
{
    // Compute sums of a batch of concatenated sequences
    int b = blockIdx.x * blockDim.x + threadIdx.x; // Batch-item we're working on
    if (b >= B) return;

    // Go to the regions we're working on
    for (int i=0; i < b; ++i) {
        output += O;
	X += lengths[i] * O;
    }
    int length = lengths[b];
    // Each invocation of the kernel sums one batch.
    for (int _=0; _ < length; ++_) // Iterate over rows
    {
        for (int i=0; i < O; ++i) 
        {
          output[i] += X[i];
        }
        X += O;
    }
}


extern "C" __global__
void maxout(float* best, int* which,
        const float* cands, int B, int O, int P)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x; 
    if (b >= B) return;

    // Go to the regions we're working on
    for (int i=0; i < b; ++i) {
        best += O;
        which += O;
        cands += O * P;
    }

    for (int i=0; i < O; ++i)
    {
        which[i] = 0
        best[i] = cands[0];
        for (int p=1; p < P; ++p)
	{
            if (cands[i+p] > best[0])
	    {
                which[i] = p;
                best[i] = cands[i+p];
	    }
	}
    }
}


extern "C" __global__
void max_pool(float* maxes, int* which,
    const float* X, const int* lengths, int B, int T, int O)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x; // Batch-item we're working on
    if (b >= B) return;

    // Go to the regions we're working on
    for (int i=0; i < b; ++i) {
        maxes += O;
        which += O;
	X += lengths[i] * O;
    }
 
    // Each invocation of the kernel maxes one batch.
    // Start by assuming maxes are at i=0
    for (int j=0; j < O; ++j) {
        maxes[j] = X[j];
	which[j] = 0;
    }
    X += O;
    
    int length = lengths[b];
    for (int i=1; i < length; ++i) // Iterate over rows
    {
        for (int j=0; j < O; ++j)
	{
            if (X[j] > maxes[j])
            {
                maxes[j] = X[j];
                which[j] = i;
	    }
	}
	X += O;
    }
}
